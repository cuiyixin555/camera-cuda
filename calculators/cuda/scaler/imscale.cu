#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024-2025 Cui, Xin

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//

/**
 * @brief Scaling image kernel function for YUV420P with bilinear
 * interpolation
 * f(x,y) = f(0,0)(1-x)(1-y) + f(1,0)x(1-y) + f(0,1)(1-x)y + f(1,1)xy;
 *
 * @param pInYData input nv12 image for Y planer
 * @param pInUData input nv12 image for U planer
 * @param pInVData input nv12 image for V planer
 * @param pInWidth input image width
 * @param pInHeight input image height
 * @param pOutYData output image for Y planer
 * @param pOutUData output image for U planer
 * @param pOutVData output image for V planer
 * @param pOutWidth output image width
 * @param pOutHeight output image height
 */
__global__ void ReSizeKernel_Bilinear_YUV420P(
    unsigned char *pInYData, unsigned char *pInUData, unsigned char *pInVData,
    int pInWidth, int pInHeight, unsigned char *pOutYData,
    unsigned char *pOutUData, unsigned char *pOutVData, int pOutWidth,
    int pOutHeight) {
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  int tidy = threadIdx.y + blockDim.y * blockIdx.y;

  if (tidx < pOutWidth && tidy < pOutHeight) {
    float srcX = tidx * ((float)(pInWidth - 1) / (pOutWidth - 1));
    float srcY = tidy * ((float)(pInHeight - 1) / (pOutHeight - 1));

    // 计算取图像坐标
    int fx0 = srcX;
    int fy0 = srcY;
    int fx1 = srcX > fx0 ? fx0 + 1 : fx0;
    int fy1 = srcY > fy0 ? fy0 + 1 : fy0;

    // 计算取像素比例
    float xProportion = srcX - fx0;
    float yProportion = srcY - fy0;

    // 四个输入坐标
    int idx_in_y_00 = fy0 * pInWidth + fx0;
    int idx_in_uv_00 = fy0 / 2 * pInWidth / 2 + fx0 / 2;

    int idx_in_y_10 = fy1 * pInWidth + fx0;
    int idx_in_uv_10 = fy1 / 2 * pInWidth / 2 + fx0 / 2;

    int idx_in_y_01 = fy0 * pInWidth + fx1;
    int idx_in_uv_01 = fy0 / 2 * pInWidth / 2 + fx1 / 2;

    int idx_in_y_11 = fy1 * pInWidth + fx1;
    int idx_in_uv_11 = fy1 / 2 * pInWidth / 2 + fx1 / 2;

    // 输出坐标
    int idx_out_y = tidy * pOutWidth + tidx;
    int idx_out_uv = tidy / 2 * pOutWidth / 2 + tidx / 2;

    // Y
    pOutYData[idx_out_y] =
        pInYData[idx_in_y_00] * (1 - xProportion) * (1 - yProportion) +
        pInYData[idx_in_y_10] * xProportion * (1 - yProportion) +
        pInYData[idx_in_y_01] * (1 - xProportion) * yProportion +
        pInYData[idx_in_y_11] * xProportion * yProportion;

    // U
    pOutUData[idx_out_uv] =
        pInUData[idx_in_uv_00] * (1 - xProportion) * (1 - yProportion) +
        pInUData[idx_in_uv_10] * xProportion * (1 - yProportion) +
        pInUData[idx_in_uv_01] * (1 - xProportion) * yProportion +
        pInUData[idx_in_uv_11] * xProportion * yProportion;

    // V
    pOutVData[idx_out_uv] =
        pInVData[idx_in_uv_00] * (1 - xProportion) * (1 - yProportion) +
        pInVData[idx_in_uv_10] * xProportion * (1 - yProportion) +
        pInVData[idx_in_uv_01] * (1 - xProportion) * yProportion +
        pInVData[idx_in_uv_11] * xProportion * yProportion;
  }
}

/**
 * @brief Modify size with bilinear interpolation YUV420P
 *
 * @param frame input image
 * @param width input width
 * @param height input height
 * @return AVFrame*
 */
AVFrame *ReSize_Bilinear_YUV420P(AVFrame *frame, int width, int height) {
  auto img_size_y = width * height * sizeof(unsigned char);
  auto img_size_uv = (width / 2) * (height / 2) * sizeof(unsigned char);

  AVFrame *dstImg;
  unsigned char *outputY = nullptr;
  unsigned char *outputU = nullptr;
  unsigned char *outputV = nullptr;

  dstImg = av_frame_alloc();
  av_image_alloc(dstImg->data, dstImg->linesize, width, height,
                 (AVPixelFormat)frame->format, 1);
  dstImg->width = width;
  dstImg->height = height;
  dstImg->format = (AVPixelFormat)frame->format;

  hipMalloc(&outputY, img_size_y);
  hipMalloc(&outputU, img_size_uv);
  hipMalloc(&outputV, img_size_uv);

  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
  ReSizeKernel_Bilinear_YUV420P<<<grid, block>>>(
      frame->data[0], frame->data[1], frame->data[2], frame->width,
      frame->height, outputY, outputU, outputV, width, height);
  hipDeviceSynchronize();

  // 图像从 Gpu 拷贝到 Cpu
  hipMemcpy(dstImg->data[0], outputY, img_size_y, hipMemcpyDeviceToHost);
  hipMemcpy(dstImg->data[1], outputU, img_size_uv, hipMemcpyDeviceToHost);
  hipMemcpy(dstImg->data[2], outputV, img_size_uv, hipMemcpyDeviceToHost);
  return dstImg;
}

/**
 * @brief Scaling image kernel function NV12
 * Bilinear interpolation
 * f(x,y) = f(0,0)(1-x)(1-y) + f(1,0)x(1-y) + f(0,1)(1-x)y + f(1,1)xy;
 *
 * @param pInYData input image for Y planer
 * @param pInUVData input image for UV planer
 * @param pInWidth  input image width
 * @param pInHeight input image height
 * @param pOutYData output image for Y planer
 * @param pOutUVData output image for UV planer
 * @param pOutWidth output image width
 * @param pOutHeight output image height
 */
__global__ void ReSizeKernel_Bilinear_NV12(unsigned char *pInYData,
                                           unsigned char *pInUVData,
                                           int pInWidth, int pInHeight,
                                           unsigned char *pOutYData,
                                           unsigned char *pOutUVData,
                                           int pOutWidth, int pOutHeight) {
  int tidx = threadIdx.x + blockDim.x * blockIdx.x;
  int tidy = threadIdx.y + blockDim.y * blockIdx.y;

  if (tidx < pOutWidth && tidy < pOutHeight) {
    float srcX = tidx * ((float)(pInWidth - 1) / (pOutWidth - 1));
    float srcY = tidy * ((float)(pInHeight - 1) / (pOutHeight - 1));

    /// calculate image coordinates
    int fx0 = srcX;
    int fy0 = srcY;
    int fx1 = srcX > fx0 ? fx0 + 1 : fx0;
    int fy1 = srcY > fy0 ? fy0 + 1 : fy0;

    /// calculate pixel ratio
    float xProportion = srcX - fx0;
    float yProportion = srcY - fy0;

    /// four input coordinates
    int idx_in_y_00 = fy0 * pInWidth + fx0;
    int idx_in_uv_00 = fy0 / 2 * pInWidth + fx0;

    int idx_in_y_10 = fy1 * pInWidth + fx0;
    int idx_in_uv_10 = fy1 / 2 * pInWidth + fx0;

    int idx_in_y_01 = fy0 * pInWidth + fx1;
    int idx_in_uv_01 = fy0 / 2 * pInWidth + fx1;

    int idx_in_y_11 = fy1 * pInWidth + fx1;
    int idx_in_uv_11 = fy1 / 2 * pInWidth + fx1;

    /// output coordinates
    int idx_out_y = tidy * pOutWidth + tidx;
    int idx_out_uv = tidy / 2 * pOutWidth + tidx;

    // Y
    pOutYData[idx_out_y] =
        pInYData[idx_in_y_00] * (1 - xProportion) * (1 - yProportion) +
        pInYData[idx_in_y_10] * xProportion * (1 - yProportion) +
        pInYData[idx_in_y_01] * (1 - xProportion) * yProportion +
        pInYData[idx_in_y_11] * xProportion * yProportion;

    // U
    pOutUVData[tidx % 2 == 0 ? idx_out_uv : idx_out_uv - 1] =
        pInUVData[fx0 % 2 == 0 ? idx_in_uv_00 : idx_in_uv_00 - 1] *
            (1 - xProportion) * (1 - yProportion) +
        pInUVData[fx0 % 2 == 0 ? idx_in_uv_10 : idx_in_uv_10 - 1] *
            xProportion * (1 - yProportion) +
        pInUVData[fx1 % 2 == 0 ? idx_in_uv_01 : idx_in_uv_01 - 1] *
            (1 - xProportion) * yProportion +
        pInUVData[fx1 % 2 == 0 ? idx_in_uv_11 : idx_in_uv_11 - 1] *
            xProportion * yProportion;

    // V
    pOutUVData[tidx % 2 == 0 ? idx_out_uv + 1 : idx_out_uv] =
        pInUVData[fx0 % 2 == 0 ? idx_in_uv_00 + 1 : idx_in_uv_00] *
            (1 - xProportion) * (1 - yProportion) +
        pInUVData[fx0 % 2 == 0 ? idx_in_uv_10 + 1 : idx_in_uv_10] *
            xProportion * (1 - yProportion) +
        pInUVData[fx1 % 2 == 0 ? idx_in_uv_01 + 1 : idx_in_uv_01] *
            (1 - xProportion) * yProportion +
        pInUVData[fx1 % 2 == 0 ? idx_in_uv_11 + 1 : idx_in_uv_11] *
            xProportion * yProportion;
  }
}

/**
 * @brief Modify size bilinear interpolation NV12
 *
 * @param frame input image
 * @param width input image width
 * @param height input image height
 * @return AVFrame*
 */
AVFrame *ReSize_Bilinear_NV12(AVFrame *frame, int width, int height) {
  auto img_size_y = width * height * sizeof(unsigned char);
  auto img_size_uv = width * (height / 2) * sizeof(unsigned char);

  AVFrame *dstImg;
  unsigned char *outputY = nullptr;
  unsigned char *outputUV = nullptr;

  dstImg = av_frame_alloc();
  av_image_alloc(dstImg->data, dstImg->linesize, width, height,
                 (AVPixelFormat)frame->format, 1);
  dstImg->width = width;
  dstImg->height = height;
  dstImg->format = (AVPixelFormat)frame->format;

  hipMalloc(&outputY, img_size_y);
  hipMalloc(&outputUV, img_size_uv);

  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
  ReSizeKernel_Bilinear_NV12<<<grid, block>>>(frame->data[0], frame->data[1],
                                              frame->width, frame->height,
                                              outputY, outputUV, width, height);
  hipDeviceSynchronize();

  /// Image copy from GPU to CPU
  hipMemcpy(dstImg->data[0], outputY, img_size_y, hipMemcpyDeviceToHost);
  hipMemcpy(dstImg->data[1], outputUV, img_size_uv, hipMemcpyDeviceToHost);
  return dstImg;
}