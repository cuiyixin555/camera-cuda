#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024-2025 Cui, Xin

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//

#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <string>
#include <vector>

using namespace cv;
using namespace std;

__global__ void resizeGPUNearest(const unsigned char *src, int srcWidth,
                                 int srcHeight, unsigned char *dst,
                                 int dstWidth, int dstHeight) {
  //核函数会在每个thread上运行，这里求的x、y是当前thread的坐标，同时也代表当前要处理的像素的坐标
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x >= dstWidth || y >= dstHeight)
    return;
  //以指针的形式操作图像，outPosition是指目标图像素在内存中的位置
  int outPosition = y * dstWidth + x;
  //求取对应原图的像素点，srcPosition是指原图像素在内存中的位置
  int srcX = x * srcWidth /
             dstWidth; //如果出现浮点数，这里就会向下取整，以此来表示最近邻
  int srcY =
      y * srcHeight / dstHeight; //（如果不喜欢向下取整，也可以选择四舍五入）
  int srcPosition = srcY * srcWidth + srcX;
  //为目标图像素赋值。RGB三通道，在内存中的位置是挨着的。
  dst[outPosition * 3 + 0] = src[srcPosition * 3 + 0];
  dst[outPosition * 3 + 1] = src[srcPosition * 3 + 1];
  dst[outPosition * 3 + 2] = src[srcPosition * 3 + 2];
}

__global__ void resizeGPUBilinear(const unsigned char *src, int srcWidth,
                                  int srcHeight, unsigned char *dst,
                                  int dstWidth, int dstHeight) {
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x >= dstWidth || y >= dstHeight)
    return;
  int dstOffset = y * dstWidth + x; //目标图像素在内存中的位置
  //根据缩放比例，计算在原图的坐标（浮点值）
  int srcXf = x * ((float)srcWidth / dstWidth);
  int srcYf = y * ((float)srcHeight / dstHeight);
  //向下取整，得到四个像素中，左上的像素坐标
  int srcX = (int)srcXf;
  int srcY = (int)srcYf;
  // u就是上面算法中的x-x1，1-u就是x2-x
  int u = srcXf - srcX;
  int v = srcYf - srcY;

  // P=(Q11)(x2- x)(y2-y) + (Q21)(x- x1)(y2- y) + (Q12)(x2- x)(- y1) + (Q22)(x -
  // x1)(y- y1)
  dst[dstOffset] = 0;
  dst[dstOffset] += (1 - u) * (1 - v) * src[(srcY * srcWidth + srcX)];
  dst[dstOffset] += (1 - u) * v * src[((srcY + 1) * srcWidth + srcX)];
  dst[dstOffset] += u * (1 - v) * src[(srcY * srcWidth + srcX + 1)];
  dst[dstOffset] += u * v * src[((srcY + 1) * srcWidth + srcX + 1)];
  //(srcY+1)*srcWidth+srcX+1)是右下角的像素点在内存中的位置
}

//主函数和上一个算法代码一样，唯一区别就是，为了代码简单，把图片变成了灰度图
void resizeBilinear() {
  Mat src = imread("data\\image\\house_256x256.png", 0);
  int srcWidth = src.cols;
  int srcHeight = src.rows;
  int dstWidth = 512;
  int dstHeight = 512;

  unsigned char *devSrc;
  unsigned char *devDst;

  hipMalloc((void **)&devSrc, srcWidth * srcHeight * sizeof(unsigned char));
  hipMalloc((void **)&devDst, dstWidth * dstHeight * sizeof(unsigned char));
  hipMemcpy(devSrc, src.data, srcWidth * srcHeight * sizeof(unsigned char),
             hipMemcpyHostToDevice);

  dim3 blocks((dstWidth + 15) / 16, (dstHeight + 15) / 16);
  dim3 threads(16, 16);
  resizeGPUBilinear<<<blocks, threads>>>(devSrc, srcWidth, srcHeight, devDst,
                                         dstWidth, dstHeight);

  Mat dst(Size(dstWidth, dstHeight), CV_8UC1);
  hipMemcpy(dst.data, devDst, dstWidth * dstHeight * sizeof(unsigned char),
             hipMemcpyDeviceToHost);

  vector<int> comprocession_params;
  comprocession_params.push_back(IMWRITE_PNG_COMPRESSION);
  comprocession_params.push_back(9);
  imwrite("resize_bilinear.png", dst, comprocession_params);
  hipFree(devSrc);
  hipFree(devDst);
}

void resizeNearest() {
  Mat src = imread("data\\image\\house_512x512.png"); //使用opencv
  int srcWidth = src.cols;
  int srcHeight = src.rows;
  int dstWidth = 256; //目标图的大小
  int dstHeight = 256;

  unsigned char *devSrc;
  unsigned char *devDst;

  //在GPU上为两张图申请存储空间
  hipMalloc((void **)&devSrc,
             srcWidth * srcHeight * 3 * sizeof(unsigned char));
  hipMalloc((void **)&devDst,
             dstWidth * dstHeight * 3 * sizeof(unsigned char));
  //把原图复制到GPU上，注意图片数据格式的变化
  hipMemcpy(devSrc, (unsigned char *)(src.data),
             srcWidth * srcHeight * 3 * sizeof(unsigned char),
             hipMemcpyHostToDevice);

  dim3 blocks((dstWidth + 15) / 16, (dstHeight + 15) / 16);
  dim3 threads(16, 16);
  //调用核函数，重点关注blocks与threads的设置，这样设置是为了让thread的坐标代表目标图像素的坐标
  resizeGPUNearest<<<blocks, threads>>>(devSrc, srcWidth, srcHeight, devDst,
                                        dstWidth, dstHeight);
  //将处理完的目标图拷贝回来
  Mat dst(Size(dstWidth, dstHeight), CV_8UC3);
  hipMemcpy(dst.data, devDst, dstWidth * dstHeight * 3 * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  //使用opencv保存新图片
  vector<int> comprocession_params;
  comprocession_params.push_back(IMWRITE_PNG_COMPRESSION);
  comprocession_params.push_back(9);
  imwrite("resize_nearest.png", dst, comprocession_params);
  hipFree(devSrc);
  hipFree(devDst);
}
